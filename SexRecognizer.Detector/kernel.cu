#ifndef __HIPCC__  
#define __HIPCC__
#endif

#include "hip/hip_runtime.h"
#include ""
#include "../SexRecognizer.Common/DirectoryLoader.h"
#include "../SexRecognizer.Common/MatrixHelpers.h"
#include "SilhouetteExtractor.h"
#include "ImageResizer.h"
#include "BackgroundSubtractor.h"
///////////////////////////////////////////////////////////////

//opencv
#include "opencv2/imgcodecs.hpp"
#include "opencv2/imgproc.hpp"
#include "opencv2/videoio.hpp"
#include "opencv2/highgui.hpp"
#include "opencv2/video.hpp"
#include "opencv2/core.hpp"
//C
#include <stdio.h>
//C++
#include <iostream>
#include <sstream>
#include <math.h>

#define MOG2_SUBTRACTION 0

using namespace cv;
using namespace std;
using namespace cv::cuda;
using namespace Extract;


void help();
void help()
{
	cout
		<< "--------------------------------------------------------------------------" << endl
		<< "This program will resize the and extract human from given images" << endl
		<< "using OpenCV. You can process only image sequences. The path should point to first img." << endl
		<< endl
		<< "Usage:" << endl
		<< "./bg_sub <image filename>" << endl
		<< "for example: ./bg_sub /data/images/1.png" << endl
		<< "--------------------------------------------------------------------------" << endl
		<< endl;
}
int main(int argc, char* argv[])
{
	//print help information
	help();
	//check for the input parameter correctness
	if (argc != 2) {
		cerr << "Incorret input list" << endl;
		cerr << "exiting..." << endl;
		return EXIT_FAILURE;
	}
	
	Common::DirectoryLoader dirLoader(argv[1]); // load directory
	std::vector<cv::Mat> frames = dirLoader.GetFrames();
	Extract::ImageResizer resizer(320, 240);  // resize frames
	std::vector<cv::Mat> resizedFrames = resizer.resizeFrames(frames);

	Extract::SilhouetteExtractor extractor(MOG2_SUBTRACTION);
	
	std::vector<int> xs = extractor.extract(resizedFrames); //extract data
	
	return 0;
}